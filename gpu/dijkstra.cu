
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
using namespace std;
#define f(i,a,b) for(int i=a;i<b;++i)
 
typedef struct edge{
    int u, v, w;
}edge;

float exec_time = 0;
 
bool edgeComparator(edge &e1, edge &e2){
    if(e1.u==e2.u)return e1.v<e2.v;
    return e1.u<e2.u;
}
 
__global__
void initialise(int n, int *wt, bool *mask)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride){
        mask[i]=0;
        if(!i){
            wt[i]=0;
        }
        else{
            wt[i]=1000000000;
        }
    }   
}
 
__global__
void threshold(int n, int* v, int *wt, int *ea, int *ew, bool *mask, int *thrd){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride){
        //printf("id %d\n", i);
        if(!mask[i]&&wt[i]<1000000000){
            for(int j=v[i];j<v[i+1];++j)if(!mask[ea[j]]){
                //printf("edge %d %d\n", i, ea[j]);
                /*if(*thrd > wt[i] + ew[j]){
                    atomicCAS(thrd, *thrd, wt[i]+ew[j]);
                    //printf("%d %d\n", ea[j], *thrd);
                }*/
                atomicMin(thrd, wt[i]+ew[j]);
            }
        }
    }
}
 
__global__
void relax(int n, int *v, int *wt, int *ea, int *ew, bool *mask, int *thrd){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride){
        if(!mask[i]&&wt[i]<*thrd){
            mask[i]=1;
            for(int j=v[i];j<v[i+1];++j){
                //if(wt[ea[j]] > wt[i] + ew[j])atomicCAS(&wt[ea[j]], wt[ea[j]], wt[i]+ew[j]);
                atomicMin(&wt[ea[j]], wt[i]+ew[j]);
            }
        }
    }
}
 
void dijkstra(int n, int *v, int *wt, int *ea, int *ew, bool *mask, int *thrd){
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    //cout<<"s1"<<endl;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    initialise<<<numBlocks, blockSize>>>(n, wt, mask);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float tmp = 0;
    hipEventElapsedTime(&tmp, start, stop);
    exec_time+=tmp;
    //cout<<"s2"<<endl;
    *thrd = 0;
    //cout<<"bf"<<endl;
    //cout<<*thrd<<endl;
    //cout<<"bf2"<<endl;
    while(*thrd<1000000000){
        *thrd=1000000000;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
        threshold<<<numBlocks, blockSize>>>(n, v, wt, ea, ew, mask, thrd);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        tmp = 0;
        hipEventElapsedTime(&tmp, start, stop);
        exec_time+=tmp;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
        relax<<<numBlocks, blockSize>>>(n, v, wt, ea, ew, mask, thrd);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        tmp = 0;
        hipEventElapsedTime(&tmp, start, stop);
        exec_time+=tmp;
    }
    f(i,0,n)cout<<wt[i]<<" ";cout<<endl;
}
 
 
int main(void)
{
    ios_base::sync_with_stdio(false);
    cin.tie(0);
    cout.tie(0);
    int n, m, l, r, w;
    cin>>n>>m;
    vector<edge> edges;
    f(i,0,m)cin>>l>>r>>w, edges.push_back({l-1, r-1, w});
    sort(edges.begin(), edges.end(), edgeComparator);
    
    int *wt, *v, *ea, *ew, *thrd;
    bool *mask;
 
    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&wt, n*sizeof(int));
    hipMallocManaged(&v, (n+1)*sizeof(int));
    hipMallocManaged(&ea, m*sizeof(int));
    hipMallocManaged(&ew, m*sizeof(int));
    hipMallocManaged(&mask, n*sizeof(bool));
    hipMallocManaged(&thrd, sizeof(int));
    
    int curv=0;v[0]=0;
    f(i,0,m){
        while(curv<edges[i].u)v[++curv]=i;
        ea[i]=edges[i].v;
        ew[i]=edges[i].w;
    }
    while(curv<n)v[++curv]=m;
    /*f(i,0,n)cout<<v[i]<<" ";cout<<endl;
    f(i,0,m)cout<<ea[i]<<" ";cout<<endl;
    f(i,0,m)cout<<ew[i]<<" ";cout<<endl;*/
 
    dijkstra(n, v, wt, ea, ew, mask, thrd);
 
    // Free memory
    hipFree(wt);
    hipFree(v);
    hipFree(ea);
    hipFree(ew);
    hipFree(mask);
    hipFree(thrd);
    
    ofstream out("kernel_time.txt");
    out<<"Total kernel time : "<<exec_time<<"\n";
    out.close();
 
    return 0;
}
