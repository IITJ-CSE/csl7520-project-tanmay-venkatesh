
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
using namespace std;
using namespace std::chrono;

#define f(i,a,b) for(int i=a;i<b;++i)
 
typedef struct edge{
    int u, v, w;
}edge;


const int blockSize = 256;
int numBlocks;
float exec_time = 0;
 
bool edgeComparator(edge &e1, edge &e2){
    if(e1.u==e2.u)return e1.v<e2.v;
    return e1.u<e2.u;
}
 
__global__
void initialise(int n, int *wt, bool *mask)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride){
        mask[i]=0;
        if(!i){
            wt[i]=0;
        }
        else{
            wt[i]=1000000000;
        }
    }   
}

__global__
void relax(int n, int *v, int *wt, int *ea, int *es, int *ew, bool *mask){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < v[n]; i += stride){
        if(wt[ea[i]]>wt[es[i]]+ew[i]){
            atomicExch(&wt[ea[i]], wt[es[i]]+ew[i]);
        }
    }
}

void bellman_ford(int n, int *v, int *wt, int *ea, int *es, int *ew, bool *mask){
        int numBlocks = (n + blockSize - 1) / blockSize;
        hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    initialise<<<numBlocks, blockSize>>>(n, wt, mask);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float tmp = 0;
    hipEventElapsedTime(&tmp, start, stop);
    exec_time+=tmp;
        int numBlocksE = (v[n] + blockSize - 1) / blockSize;
        for(int i=0;i<n-1;++i){
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
        relax<<<numBlocksE, blockSize>>>(n, v, wt, ea, es, ew, mask);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float tmp = 0;
        hipEventElapsedTime(&tmp, start, stop);
        exec_time+=tmp;
    }
        for(int i=0;i<n;++i)cout<<wt[i]<<" ";cout<<"\n";
}
 

 
 
int main(void)
{
    ios_base::sync_with_stdio(false);
    cin.tie(0);
    cout.tie(0);
    int n, m, l, r, w;
    cin>>n>>m;
    vector<edge> edges;
    f(i,0,m)cin>>l>>r>>w, edges.push_back({l-1, r-1, w});
    sort(edges.begin(), edges.end(), edgeComparator);
    
    int *wt, *v, *ea, *es, *ew;
    bool *mask;
    auto start = high_resolution_clock::now();
    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&wt, n*sizeof(int));
    hipMallocManaged(&v, (n+1)*sizeof(int));
    hipMallocManaged(&ea, m*sizeof(int));
        hipMallocManaged(&es, m*sizeof(int));
    hipMallocManaged(&ew, m*sizeof(int));
    hipMallocManaged(&mask, n*sizeof(bool));
    
    int curv=0;v[0]=0;
    f(i,0,m){
        while(curv<edges[i].u)v[++curv]=i;
        ea[i]=edges[i].v;
        es[i]=edges[i].u;
        ew[i]=edges[i].w;
    }
    while(curv<n)v[++curv]=m;
    /*f(i,0,n)cout<<v[i]<<" ";cout<<endl;
    f(i,0,m)cout<<ea[i]<<" ";cout<<endl;
    f(i,0,m)cout<<ew[i]<<" ";cout<<endl;*/
 
    bellman_ford(n, v, wt, ea, es, ew, mask);
 
    // Free memory
    hipFree(wt);
    hipFree(v);
    hipFree(ea);
        hipFree(es);
    hipFree(ew);
    hipFree(mask);
    
    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(stop - start);
    ofstream ot("gpu_perf.txt");
    ot<<duration.count()/1000.0<<"\n";
 
    return 0;
}
