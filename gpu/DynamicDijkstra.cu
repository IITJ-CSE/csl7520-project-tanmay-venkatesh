
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
using namespace std;
#define f(i,a,b) for(int i=a;i<b;++i)
 
typedef struct edge{
    int u, v, w;
}edge;
 
const int blockSize = 256;
int numBlocks;
float exec_time = 0;
 
bool edgeComparator(edge &e1, edge &e2){
    if(e1.u==e2.u)return e1.v<e2.v;
    return e1.u<e2.u;
}
 
__global__
void initialise(int n, int *wt, bool *mask)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride){
        mask[i]=0;
        if(!i){
            wt[i]=0;
        }
        else{
            wt[i]=1000000000;
        }
    }
}
 
__global__
void threshold(int n, int* v, int *wt, int *ea, int *ew, bool *mask, int *thrd){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride){
        //printf("id %d\n", i);
        if(!mask[i]&&wt[i]<1000000000){
            for(int j=v[i];j<v[i+1];++j)if(!mask[ea[j]]){
                //printf("edge %d %d\n", i, ea[j]);
                /*if(*thrd > wt[i] + ew[j]){
                    atomicCAS(thrd, *thrd, wt[i]+ew[j]);
                    //printf("%d %d\n", ea[j], *thrd);
                }*/
               atomicMin(thrd, wt[i]+ew[j]);
            }
        }
    }
}
 
__global__
void relax(int n, int *v, int *wt, int *ea, int *ew, bool *mask, int *thrd){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride){
        if(!mask[i]&&wt[i]<*thrd){
            mask[i]=1;
            for(int j=v[i];j<v[i+1];++j){
                //if(wt[ea[j]] > wt[i] + ew[j])atomicCAS(&wt[ea[j]], wt[ea[j]], wt[i]+ew[j]);
                atomicMin(&wt[ea[j]], wt[i]+ew[j]);
            }
        }
    }
}
 
void dijkstra(int n, int *v, int *wt, int *ea, int *ew, bool *mask, int *thrd){
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    initialise<<<numBlocks, blockSize>>>(n, wt, mask);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float tmp = 0;
    hipEventElapsedTime(&tmp, start, stop);
    exec_time+=tmp;
    *thrd = 0;
    while(*thrd<1000000000){
        *thrd=1000000000;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
        threshold<<<numBlocks, blockSize>>>(n, v, wt, ea, ew, mask, thrd);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        tmp = 0;
        hipEventElapsedTime(&tmp, start, stop);
        exec_time+=tmp;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
        relax<<<numBlocks, blockSize>>>(n, v, wt, ea, ew, mask, thrd);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        tmp = 0;
        hipEventElapsedTime(&tmp, start, stop);
        exec_time+=tmp;
    }
    f(i,0,n)cout<<wt[i]<<" ";cout<<endl;
}
 
__global__
void update_v(int n, int l, int inc, int *v){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i <= n; i += stride){
        if(i>l)v[i]+=inc;
    }
}
 
__global__
void store_ea_es_and_ew(int n, int idx, int *v, int *ea, int *es, int *ew, int *tmp_ea, int *tmp_es, int *tmp_ew){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < v[n]; i += stride){
        if(i>idx){
            tmp_ea[i]=ea[i];
            tmp_es[i]=es[i];
            tmp_ew[i]=ew[i];
        }
    }
}
 
__global__
void update_ea_es_and_ew(int n, int idx, int gap, int *v, int *ea, int *es, int *ew, int *tmp_ea, int *tmp_es, int *tmp_ew){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < v[n]; i += stride){
        if(i>=idx){
            ea[i]=tmp_ea[i+gap];
            es[i]=tmp_es[i+gap];
            ew[i]=tmp_ew[i+gap];
        }
    }
}
 
__global__
void find_edge_index(int n, int l, int r, int *v, int *es, int *ea, int *edge_idx){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < v[n]; i += stride){
        //printf("%d %d\n",es[i],ea[i]);
        if(es[i]==l&&ea[i]==r)*edge_idx=i;
    }
}
 
void update_graph(int n, int l, int r, int w, int *v, int *ea, int *es, int *ew, int *tmp_ea, int *tmp_es, int *tmp_ew){
    int *edge_idx;
    hipMallocManaged(&edge_idx, sizeof(int));
    *edge_idx=-1;
    int numBlocksE = (v[n] + blockSize - 1) / blockSize;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    find_edge_index<<<numBlocksE, blockSize>>>(n, l, r, v, es, ea, edge_idx);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float tmp = 0;
    hipEventElapsedTime(&tmp, start, stop);
    exec_time+=tmp;
    int idx=*edge_idx;
    hipFree(edge_idx);
    //cout<<idx<<endl;
    if(idx!=-1){
        if(w!=-1)ew[idx]=w;
        else{
            hipEventCreate(&start);
            hipEventCreate(&stop);
            hipEventRecord(start);
            store_ea_es_and_ew<<<numBlocksE, blockSize>>>(n, idx, v, ea, es, ew, tmp_ea, tmp_es, tmp_ew);
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            tmp = 0;
            hipEventElapsedTime(&tmp, start, stop);
            exec_time+=tmp;
            hipEventCreate(&start);
            hipEventCreate(&stop);
            hipEventRecord(start);
            update_v<<<numBlocks+1, blockSize>>>(n, l, -1, v);
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            tmp = 0;
            hipEventElapsedTime(&tmp, start, stop);
            exec_time+=tmp;
            hipEventCreate(&start);
            hipEventCreate(&stop);
            hipEventRecord(start);
            update_ea_es_and_ew<<<numBlocksE, blockSize>>>(n, idx, 1, v, ea, es, ew, tmp_ea, tmp_es, tmp_ew);
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            tmp = 0;
            hipEventElapsedTime(&tmp, start, stop);
            exec_time+=tmp;
        }
    }
    else{
        numBlocksE = v[n]/ blockSize + 1;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
        store_ea_es_and_ew<<<numBlocksE, blockSize>>>(n, v[l+1]-1, v, ea, es, ew, tmp_ea, tmp_es, tmp_ew);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        tmp = 0;
        hipEventElapsedTime(&tmp, start, stop);
        exec_time+=tmp;
        es[v[l+1]]=l;
        ea[v[l+1]]=r;
        ew[v[l+1]]=w;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
        update_v<<<numBlocks+1, blockSize>>>(n, l, 1, v);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        tmp = 0;
        hipEventElapsedTime(&tmp, start, stop);
        exec_time+=tmp;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
        update_ea_es_and_ew<<<numBlocksE, blockSize>>>(n, v[l+1], -1, v, ea, es, ew, tmp_ea, tmp_es, tmp_ew);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        tmp = 0;
        hipEventElapsedTime(&tmp, start, stop);
        exec_time+=tmp;
    }
}
 
void process_queries(int n, int q, int *v, int *wt, int *ea, int* es, int *ew, bool *mask, int *thrd, int *tmp_ea, int *tmp_es, int *tmp_ew){
    while(q--){
        int e, l, r, w=-1;
        cin>>e>>l>>r;
        l--;
        r--;
        if(e)cin>>w;
        update_graph(n, l, r, w, v, ea, es, ew, tmp_ea, tmp_es, tmp_ew);
        dijkstra(n, v, wt, ea, ew, mask, thrd);
    }
}
 
int main(void)
{
    ios_base::sync_with_stdio(false);
    cin.tie(0);
    cout.tie(0);
    int n, m, l, r, w, q;
    cin>>n>>m>>q;
 
    numBlocks = (n + blockSize - 1) / blockSize;
 
    vector<edge> edges;
    f(i,0,m)cin>>l>>r>>w, edges.push_back({l-1, r-1, w});
    sort(edges.begin(), edges.end(), edgeComparator);
 
    int *wt, *v, *es, *ea, *ew, *thrd, *tmp_ea, *tmp_es, *tmp_ew;
    bool *mask;
 
    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&wt, n*sizeof(int));
    hipMallocManaged(&v, (n+1)*sizeof(int));
    hipMallocManaged(&es, (m+q)*sizeof(int));
    hipMallocManaged(&ea, (m+q)*sizeof(int));
    hipMallocManaged(&ew, (m+q)*sizeof(int));
    hipMallocManaged(&tmp_es, (m+q)*sizeof(int));
    hipMallocManaged(&tmp_ea, (m+q)*sizeof(int));
    hipMallocManaged(&tmp_ew, (m+q)*sizeof(int));
    hipMallocManaged(&mask, n*sizeof(bool));
    hipMallocManaged(&thrd, sizeof(int));
 
    int curv=0;v[0]=0;
    f(i,0,m){
        while(curv<edges[i].u)v[++curv]=i;
        ea[i]=edges[i].v;
        es[i]=edges[i].u;
        ew[i]=edges[i].w;
    }
    while(curv<n)v[++curv]=m;
    /*f(i,0,n)cout<<v[i]<<" ";cout<<endl;
    f(i,0,m)cout<<ea[i]<<" ";cout<<endl;
    f(i,0,m)cout<<ew[i]<<" ";cout<<endl;*/
 
    dijkstra(n, v, wt, ea, ew, mask, thrd);
    process_queries(n, q, v, wt, ea, es, ew, mask, thrd, tmp_ea, tmp_es, tmp_ew);
 
    // Free memory
    hipFree(wt);
    hipFree(v);
    hipFree(ea);
    hipFree(es);
    hipFree(ew);
    hipFree(tmp_ea);
    hipFree(tmp_es);
    hipFree(tmp_ew);
    hipFree(mask);
    hipFree(thrd);
    
    ofstream out("../kernel_perf.txt");
    out<<exec_time<<"\n";
    out.close();
 
    return 0;
}
