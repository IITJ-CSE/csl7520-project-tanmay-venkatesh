
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
using namespace std;
#define f(i, a, b) for (int i = a; i < b; ++i)

typedef struct edge
{
    int u, v, w;
} edge;

const int blockSize = 256;
int numBlocks;
float exec_time = 0;

bool edgeComparator(edge &e1, edge &e2)
{
    if (e1.u == e2.u)
        return e1.v < e2.v;
    return e1.u < e2.u;
}

__global__ void initialise(int n, int src, int ini_cost, int *wt, bool *mask)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
    {
        mask[i] = 0;
        if (i == src)
        {
            wt[i] = ini_cost;
        }
        else
        {
            wt[i] = 1000000000;
        }
    }
}

__global__ void threshold(int n, int *v, int *wt, int *ea, int *ew, bool *mask, int *thrd)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
    {
        if (!mask[i] && wt[i] < 1000000000)
        {
            for (int j = v[i]; j < v[i + 1]; ++j)
                if (!mask[ea[j]] && *thrd > wt[i] + ew[j])
                {
                    atomicExch(thrd, wt[i] + ew[j]);
                }
        }
    }
}

__global__ void relax2(int n, int *v, int *wt, int *ea, int *es, int *ew, bool *mask, int *thrd)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < v[n]; i += stride)
    {
        if (!mask[es[i]] && wt[es[i]] < *thrd)
        {
            //mask[es[i]]=1;
            if (wt[ea[i]] > wt[es[i]] + ew[i])
            {
                atomicExch(&wt[ea[i]], wt[es[i]] + ew[i]);
            }
        }
    }
    __syncthreads();
    for (int i = index; i < v[n]; i += stride)
    {
        if (!mask[es[i]] && wt[es[i]] < *thrd)
            mask[es[i]] = 1;
    }
}

__global__ void relax(int n, int *v, int *wt, int *ea, int *ew, bool *mask, int *thrd)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
    {
        if (!mask[i] && wt[i] < *thrd)
        {
            mask[i] = 1;
            for (int j = v[i]; j < v[i + 1]; ++j)
                if (wt[ea[j]] > wt[i] + ew[j])
                {
                    atomicExch(&wt[ea[j]], wt[i] + ew[j]);
                }
        }
    }
}

__global__ void find_p(int n, int *v, int *wt, int *ea, int *es, int *ew, int *p)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < v[n]; i += stride)
        if (wt[ea[i]] == wt[es[i]] + ew[i])
        {
            atomicExch(&p[ea[i]], es[i]);
        }
}

void dijkstra(int n, int *v, int *wt, int *ea, int *es, int *ew, int *p, int src = 0, int ini_cost = 0, bool find_par = 1)
{
    bool *mask;
    hipMallocManaged(&mask, n * sizeof(bool));
    int *thrd;
    hipMallocManaged(&thrd, sizeof(int));
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    initialise<<<numBlocks, blockSize>>>(n, src, ini_cost, wt, mask);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float tmp = 0;
    hipEventElapsedTime(&tmp, start, stop);
    exec_time += tmp;
    *thrd = ini_cost;
    int numBlocksE = (v[n] + blockSize - 1) / blockSize;
    while (*thrd < 1000000000)
    {
        *thrd = 1000000000;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
        threshold<<<numBlocks, blockSize>>>(n, v, wt, ea, ew, mask, thrd);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        tmp = 0;
        hipEventElapsedTime(&tmp, start, stop);
        exec_time += tmp;
        //cout<<*thrd<<"\n";
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
        relax<<<numBlocks, blockSize>>>(n, v, wt, ea, ew, mask, thrd);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        tmp = 0;
        hipEventElapsedTime(&tmp, start, stop);
        exec_time += tmp;
    }

    if (find_par)
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
        find_p<<<numBlocksE, blockSize>>>(n, v, wt, ea, es, ew, p);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        tmp = 0;
        hipEventElapsedTime(&tmp, start, stop);
        exec_time += tmp;
    }

    hipFree(mask);
    hipFree(thrd);
}

__global__ void update_v(int n, int l, int inc, int *v)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i <= n; i += stride)
    {
        if (i > l)
            v[i] += inc;
    }
}

__global__ void initialise_arr(int n, int val, int *arr)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i <= n; i += stride)
    {
        arr[i] = val;
    }
}

__global__ void update_wt(int n, int *wt, int *tmp_wt)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i <= n; i += stride)
    {
        wt[i] = min(wt[i], tmp_wt[i]);
    }
}

__global__ void store_ea_es_and_ew(int n, int idx, int *v, int *ea, int *es, int *ew, int *tmp_ea, int *tmp_es, int *tmp_ew)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < v[n]; i += stride)
    {
        if (i > idx)
        {
            tmp_ea[i] = ea[i];
            tmp_es[i] = es[i];
            tmp_ew[i] = ew[i];
        }
    }
}

__global__ void update_ea_es_and_ew(int n, int idx, int gap, int *v, int *ea, int *es, int *ew, int *tmp_ea, int *tmp_es, int *tmp_ew)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < v[n]; i += stride)
    {
        if (i >= idx)
        {
            ea[i] = tmp_ea[i + gap];
            es[i] = tmp_es[i + gap];
            ew[i] = tmp_ew[i + gap];
        }
    }
}

__global__ void find_edge_index(int n, int l, int r, int *v, int *es, int *ea, int *edge_idx)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < v[n]; i += stride)
    {
        if (es[i] == l && ea[i] == r)
            *edge_idx = i;
    }
}

void update_graph(int n, int l, int r, int w, int *v, int *ea, int *es, int *ew, int *edge_idx, bool *dec)
{
    int numBlocksE = (v[n] + blockSize - 1) / blockSize;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    find_edge_index<<<numBlocksE, blockSize>>>(n, l, r, v, es, ea, edge_idx);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float tmp = 0;
    hipEventElapsedTime(&tmp, start, stop);
    exec_time += tmp;
    int *tmp_es, *tmp_ea, *tmp_ew;
    hipMallocManaged(&tmp_es, (v[n] + 1) * sizeof(int));
    hipMallocManaged(&tmp_ea, (v[n] + 1) * sizeof(int));
    hipMallocManaged(&tmp_ew, (v[n] + 1) * sizeof(int));
    int idx = *edge_idx;
    *dec = (idx != -1 && w != -1 && w < ew[idx]);
    if (idx != -1)
    {
        if (w != -1)
            ew[idx] = w;
        else
        {
            hipEventCreate(&start);
            hipEventCreate(&stop);
            hipEventRecord(start);
            store_ea_es_and_ew<<<numBlocksE, blockSize>>>(n, idx, v, ea, es, ew, tmp_ea, tmp_es, tmp_ew);
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            tmp = 0;
            hipEventElapsedTime(&tmp, start, stop);
            exec_time += tmp;
            hipEventCreate(&start);
            hipEventCreate(&stop);
            hipEventRecord(start);
            update_v<<<numBlocks + 1, blockSize>>>(n, l, -1, v);
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            tmp = 0;
            hipEventElapsedTime(&tmp, start, stop);
            exec_time += tmp;
            hipEventCreate(&start);
            hipEventCreate(&stop);
            hipEventRecord(start);
            update_ea_es_and_ew<<<numBlocksE, blockSize>>>(n, idx, 1, v, ea, es, ew, tmp_ea, tmp_es, tmp_ew);
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            tmp = 0;
            hipEventElapsedTime(&tmp, start, stop);
            exec_time += tmp;
        }
    }
    else
    {
        numBlocksE = v[n] / blockSize + 1;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
        store_ea_es_and_ew<<<numBlocksE, blockSize>>>(n, v[l + 1] - 1, v, ea, es, ew, tmp_ea, tmp_es, tmp_ew);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        tmp = 0;
        hipEventElapsedTime(&tmp, start, stop);
        exec_time += tmp;
        es[v[l + 1]] = l;
        ea[v[l + 1]] = r;
        ew[v[l + 1]] = w;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
        update_v<<<numBlocks + 1, blockSize>>>(n, l, 1, v);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        tmp = 0;
        hipEventElapsedTime(&tmp, start, stop);
        exec_time += tmp;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
        update_ea_es_and_ew<<<numBlocksE, blockSize>>>(n, v[l + 1], -1, v, ea, es, ew, tmp_ea, tmp_es, tmp_ew);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        tmp = 0;
        hipEventElapsedTime(&tmp, start, stop);
        exec_time += tmp;
    }
    hipFree(tmp_ea);
    hipFree(tmp_es);
    hipFree(tmp_ew);
}

__global__ void find_min_d(int n, int u, int *v, int *wt, int *ea, int *es, int *ew, int *min_d)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < v[n]; i += stride)
        if (ea[i] == u && *min_d > wt[es[i]] + ew[i])
        {
            atomicExch(min_d, wt[es[i]] + ew[i]);
        }
}

__global__ void find_best_pred(int n, int u, int *v, int *wt, int *ea, int *es, int *ew, int *best_pred, int *min_d)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < v[n]; i += stride)
        if (ea[i] == u && *min_d == wt[es[i]] + ew[i])
        {
            atomicExch(best_pred, es[i]);
        }
}

__global__ void bfs(int n, int step, int *v, int *ea, int *p, int *mark)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
        if (mark[i] == -step)
        {
            for (int j = v[i]; j < v[i + 1]; ++j)
                if (!mark[j] && p[ea[j]] == i)
                {
                    mark[ea[j]] = mark[i] - 1;
                    if (!mark[n + 1])
                        mark[n + 1] = 1;
                }
        }
}

__global__ void pool_mark(int n, int val, int *mark)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i <= n; i += stride)
        if (mark[i] < 0)
        {
            mark[i] = val;
        }
}

void paint_tree(int n, int root, int val, int *v, int *ea, int *p, int *mark)
{
    hipEvent_t start, stop;
    float tmp = 0;
    mark[root] = -1;
    int step = 1;
    while (!mark[n + 1])
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
        bfs<<<numBlocks, blockSize>>>(n, step, v, ea, p, mark);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        tmp = 0;
        hipEventElapsedTime(&tmp, start, stop);
        exec_time += tmp;
        mark[n + 1] ^= 1;
        step++;
    }
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    pool_mark<<<numBlocks, blockSize>>>(n, val, mark);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    tmp = 0;
    hipEventElapsedTime(&tmp, start, stop);
    exec_time += tmp;
}

void gradual_algo(int n, int l, int r, int w, int idx, int *v, int *wt, int *ea, int *es, int *ew, int *p, bool *dec)
{
    hipEvent_t start, stop;
    float tmp;
    if (idx == -1 || *dec)
    {
        if (wt[r] < wt[l] + w)
            return;
        int *tmp_wt;
        hipMallocManaged(&tmp_wt, n * sizeof(int));
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
        initialise_arr<<<numBlocks, blockSize>>>(n, 1000000000, tmp_wt);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        tmp = 0;
        hipEventElapsedTime(&tmp, start, stop);
        exec_time += tmp;
        dijkstra(n, v, tmp_wt, ea, es, ew, p, r, wt[l] + w, 0);
        //cout<<"hggg "<<w<<endl;
        //f(i,0,n)cout<<tmp_wt[i]<<" ";cout<<endl;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
        update_wt<<<numBlocks, blockSize>>>(n, wt, tmp_wt);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        tmp = 0;
        hipEventElapsedTime(&tmp, start, stop);
        exec_time += tmp;
        hipFree(tmp_wt);
        //f(i,0,n)cout<<wt[i]<<" ";cout<<endl;
        int numBlocksE = (v[n] + blockSize - 1) / blockSize;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
        find_p<<<numBlocksE, blockSize>>>(n, v, wt, ea, es, ew, p);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        tmp = 0;
        hipEventElapsedTime(&tmp, start, stop);
        exec_time += tmp;
    }
    else
    {
        if (p[r] != l)
            return;
        int numBlocksE = (v[n] + blockSize - 1) / blockSize;
        int *best_pred, *min_d, *mark;
        hipMallocManaged(&mark, (n + 1) * sizeof(int));
        hipMallocManaged(&best_pred, sizeof(int));
        hipMallocManaged(&min_d, sizeof(int));

        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
        initialise_arr<<<numBlocks, blockSize>>>(n + 1, 0, mark);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        tmp = 0;
        hipEventElapsedTime(&tmp, start, stop);
        exec_time += tmp;
        paint_tree(n, r, 1, v, ea, p, mark);

        priority_queue<pair<int, int>, vector<pair<int, int>>, std::greater<pair<int, int>>> H;
        H.push({wt[r], r});
        while (!H.empty())
        {
            int u = H.top().second;
            H.pop();
            int old_val = wt[u];
            wt[u] = 1e9;

            // find pred_min(u)
            *best_pred = -1;
            *min_d = 1e9;
            hipEventCreate(&start);
            hipEventCreate(&stop);
            hipEventRecord(start);
            find_min_d<<<numBlocksE, blockSize>>>(n, u, v, wt, ea, es, ew, min_d);
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            tmp = 0;
            hipEventElapsedTime(&tmp, start, stop);
            exec_time += tmp;
            hipEventCreate(&start);
            hipEventCreate(&stop);
            hipEventRecord(start);
            find_best_pred<<<numBlocksE, blockSize>>>(n, u, v, wt, ea, es, ew, best_pred, min_d);
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            float tmp = 0;
            hipEventElapsedTime(&tmp, start, stop);
            exec_time += tmp;
            if (*best_pred != -1)
            {
                if (mark[*best_pred])
                    wt[u] = 1e9, H.push({wt[u], u});
                else
                {
                    mark[u] = 0;
                    wt[u] = *min_d;
                }
            }

            if (wt[u] != old_val)
            {
                for (int j = v[u]; j < v[u + 1]; ++j)
                {
                    if (p[ea[j]] == u)
                        H.push({wt[ea[j]], ea[j]});
                    else if (wt[u] + ew[j] < wt[ea[j]])
                        H.push({wt[u] + ew[j], ea[j]});
                }
            }
            else
            {
                paint_tree(n, u, 0, v, ea, p, mark);
            }
        }
        hipFree(mark);
        hipFree(best_pred);
        hipFree(min_d);
    }
}

void process_queries(int n, int q, int *v, int *wt, int *ea, int *es, int *ew, int *p)
{

    while (q--)
    {
        int e, l, r, w = -1;
        cin >> e >> l >> r;
        l--;
        r--;
        if (e)
            cin >> w;
        int *edge_idx;
        bool *dec;
        hipMallocManaged(&edge_idx, sizeof(int));
        hipMallocManaged(&dec, sizeof(bool));
        *edge_idx = -1;
        update_graph(n, l, r, w, v, ea, es, ew, edge_idx, dec);
        /*
        f(i,0,20)cout<<v[i]<<" ";cout<<"\n";
        f(i,0,20)cout<<es[i]<<" ";cout<<"\n";
        f(i,0,20)cout<<ea[i]<<" ";cout<<"\n";
        f(i,0,20)cout<<ew[i]<<" ";cout<<"\n";
        */
        int idx = *edge_idx;
        hipFree(edge_idx);
        gradual_algo(n, l, r, w, idx, v, wt, ea, es, ew, p, dec);
        hipFree(dec);
        f(i, 0, n) cout << wt[i] << " ";
        cout << "\n";
        //dijkstra(n, v, wt, ea, ew, mask, thrd);
    }
}

int main(void)
{
    ios_base::sync_with_stdio(false);
    cin.tie(0);
    cout.tie(0);
    int n, m, l, r, w, q;
    cin >> n >> m >> q;

    numBlocks = (n + blockSize - 1) / blockSize;

    vector<edge> edges;
    f(i, 0, m) cin >> l >> r >> w, edges.push_back({l - 1, r - 1, w});
    sort(edges.begin(), edges.end(), edgeComparator);

    int *wt, *p, *v, *es, *ea, *ew;

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&wt, n * sizeof(int));
    hipMallocManaged(&v, (n + 1) * sizeof(int));
    hipMallocManaged(&p, (n + 1) * sizeof(int));
    hipMallocManaged(&es, (m + q) * sizeof(int));
    hipMallocManaged(&ea, (m + q) * sizeof(int));
    hipMallocManaged(&ew, (m + q) * sizeof(int));

    int curv = 0;
    v[0] = 0;
    f(i, 0, m)
    {
        while (curv < edges[i].u)
            v[++curv] = i;
        ea[i] = edges[i].v;
        es[i] = edges[i].u;
        ew[i] = edges[i].w;
    }
    while (curv < n)
        v[++curv] = m;

    dijkstra(n, v, wt, ea, es, ew, p);
    f(i, 0, n) cout << wt[i] << " ";
    cout << endl;
    process_queries(n, q, v, wt, ea, es, ew, p);

    // Free memory
    hipFree(wt);
    hipFree(v);
    hipFree(p);
    hipFree(ea);
    hipFree(es);
    hipFree(ew);

    ofstream out("kernel_time.txt");
    out << exec_time;
    out.close();

    return 0;
}
